#include "hip/hip_runtime.h"
#include <XMath.cuh>
#include <Grid1D.h>
#include <Grid2D.h>
#include <Grid3D.h>

namespace XM
{
    void SetGridSize(std::vector<dim3> &gridBlock, const int3 &dim)
    {
		// Prepares data
		dim3 blockSize = gridBlock.back();
		dim3 gridSize;

		// Computes thread size per dimension
		gridSize.x = DivUp(dim.x, blockSize.x);
		gridSize.y = DivUp(dim.y, blockSize.y);
		gridSize.z = DivUp(dim.z, blockSize.z);

		gridBlock = { gridSize, blockSize };
    }

	void SetGridSize(std::vector<dim3>& gridBlock, const int2 &dim)
	{
		// Prepares data
		dim3 blockSize = gridBlock.back();
		dim3 gridSize;

		// Computes thread size per dimension
		gridSize.x = DivUp(dim.x, blockSize.x);
		gridSize.y = DivUp(dim.y, blockSize.y);
		gridSize.z = 1;

		gridBlock = { gridSize, blockSize };
	}

    void SetGridSize(int2 &gridBlock, const int &dim)
    {
		gridBlock.x = DivUp(dim, gridBlock.y);
    }

    int DivUp(const int &a, const int &b)
    {
		return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    int SelectCudaDevice()
    {
		int deviceCount;
		hipGetDeviceCount(&deviceCount);
		int device = deviceCount - 1;
		checkCudaErrors(hipSetDevice(device));
		return device;
    }
    
	__host__ __device__ Domain1D::Domain1D(X3::geom::Grid1D &grid)
    {
		// Main data from grid
		Xa = grid.BoundingBox().LimitsX[0];
		Xb = grid.BoundingBox().LimitsX[1];
		Dim = grid.Nx();

		// Computes Ds
		Dx = (Xb - Xa) / (float)Dim;

		// Computes inverse squared ds
		DxInvSqr = 1.f / (Dx * Dx);
    }

	__host__ __device__ Domain2D::Domain2D(X3::geom::Grid3D& grid, const bool& useXZ)
	{
		// Main data from grid
		if (useXZ)
		{
			Dim = make_int2(grid.Nx(), grid.Nz());
			A.x = grid.BoundingBox().LimitsX[0];
			A.y = grid.BoundingBox().LimitsZ[0];
			B.x = grid.BoundingBox().LimitsX[1];
			B.y = grid.BoundingBox().LimitsZ[1];
		}
		else
		{
			Dim = make_int2(grid.Nx(), grid.Ny());
			A.x = grid.BoundingBox().LimitsX[0];
			A.y = grid.BoundingBox().LimitsY[0];
			B.x = grid.BoundingBox().LimitsX[1];
			B.y = grid.BoundingBox().LimitsY[1];
		}

		// Computes Ds
		Ds = (B - A) / make_float2(Dim);

		// Computes inverse squared ds
		DsInvSqr = 1.f / (Ds * Ds);

		// Total domain size
		Size = Dim.x * Dim.y;
	}

    __host__ __device__ Domain2D::Domain2D(X3::geom::Grid2D &grid)
    {
		// Main data from grid
		Dim = make_int2(grid.Nx(), grid.Ny());
		A.x = grid.BoundingBox().LimitsX[0];
		A.y = grid.BoundingBox().LimitsY[0];
		B.x = grid.BoundingBox().LimitsX[1];
		B.y = grid.BoundingBox().LimitsY[1];

		// Computes Ds
		Ds = (B - A) / make_float2(Dim);

		// Computes inverse squared ds
		DsInvSqr = 1.f / (Ds * Ds);

		// Total domain size
		Size = Dim.x * Dim.y;
    }

	__host__ __device__ Domain3D::Domain3D(X3::geom::Grid3D& grid)
	{
		// Main data from grid
		Dim = make_int3(grid.Nx(), grid.Ny(), grid.Nz());
		A.x = grid.BoundingBox().LimitsX[0];
		A.y = grid.BoundingBox().LimitsY[0];
		A.z = grid.BoundingBox().LimitsZ[0];
		B.x = grid.BoundingBox().LimitsX[1];
		B.y = grid.BoundingBox().LimitsY[1];
		B.z = grid.BoundingBox().LimitsZ[1];

		// Computes Ds
		Ds = (B - A) / make_float3(Dim);

		// Computes inverse squared ds
		DsInvSqr = 1.f / (Ds * Ds);

		// Total domain size
		Size = Dim.x * Dim.y * Dim.z;
	}

} // namespace XMath